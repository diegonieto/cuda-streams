
#include <hip/hip_runtime.h>
/*
 * Created by Diego Nieto Muñoz
 *
 * Approach:
 * We want to compute two kernels which have two inputs and one
 * output each. I will show the benefits of using asynchronous
 * copies.
 * The architecture Fermi has three streams:
 *   -> Host To Device copies
 *   -> Kernel computation
 *   -> Device To Host copies
 *
 * Definitions:
 *   -> operation1(d1_a, d1_b, d1_c)
 *   -> operation2(d2_a, d2_b, d2_c)
 *
 *   -> 6 streams
 *   -> 4 events
 *
 * Dependencies:
 *   stream1 -> d1_a, d1_b
 *   stream2 -> operation1 | event1 -> wait for stream1
 *   stream3 -> d1_c | event2 -> wait for stream2
 *
 *   stream4 -> d2_a, d2_b
 *   stream5 -> operation2 | event3 -> wait for stream4
 *   stream6 -> d2_c | event4 -> wait for stream5 
 *
 * Flow (2 iterations):
 *   H2D          1_a, 1_b  2_a, 2_b 
 *   Computation            operation1 operation2
 *   D2H                               1_c        2_c
 */

typedef float TYPE;

// Kernel 1
__global__
void operation1(TYPE *a,
                TYPE *b,
                TYPE *c,
                int n)
{
  unsigned int i, gid = blockIdx.x*blockDim.x+threadIdx.x;
  for(i = 0; i<25; i++)
    c[gid] = sqrt(a[gid]) * sqrt(b[gid]);
}

// Kernel 2
__global__
void operation2(TYPE *a,
                TYPE *b,
                TYPE *c,
                int n)
{
  unsigned int i, gid = blockIdx.x*blockDim.x+threadIdx.x;
  for(i = 0; i<5; i++)
    c[gid] = cos(a[gid]) * sin(b[gid]);
}

// Main program
int main(int argc, char *argv[]) {

  unsigned int i, n = 1 << 20;
  unsigned int blockSize, gridSize;
  hipStream_t stream1, stream2, stream3, stream4, stream5, stream6;
  hipEvent_t e1, e2, e3, e4;

  // Host data
  TYPE *h1_a;
  TYPE *h1_b;
  TYPE *h1_c;
  TYPE *h2_a;
  TYPE *h2_b;
  TYPE *h2_c;

  // Device data
  TYPE *d1_a;
  TYPE *d1_b;
  TYPE *d1_c;
  TYPE *d2_a;
  TYPE *d2_b;
  TYPE *d2_c;

  size_t bytes = n*sizeof(TYPE);

  // Host allocation
  hipHostMalloc(&h1_a, bytes, hipHostMallocDefault);
  hipHostMalloc(&h1_b, bytes, hipHostMallocDefault);
  hipHostMalloc(&h1_c, bytes, hipHostMallocDefault);
  hipHostMalloc(&h2_a, bytes, hipHostMallocDefault);
  hipHostMalloc(&h2_b, bytes, hipHostMallocDefault);
  hipHostMalloc(&h2_c, bytes, hipHostMallocDefault);

  // Device allocation
  hipMalloc(&d1_a, bytes);
  hipMalloc(&d1_b, bytes);
  hipMalloc(&d1_c, bytes);
  hipMalloc(&d2_a, bytes);
  hipMalloc(&d2_b, bytes);
  hipMalloc(&d2_c, bytes);

  // Dimensions
  blockSize = 64;
  gridSize = (n + blockSize - 1)/blockSize;

  // Streams
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);
  hipStreamCreate(&stream4);
  hipStreamCreate(&stream5);
  hipStreamCreate(&stream6);

  // Events
  hipEventCreate(&e1);
  hipEventCreate(&e2);
  hipEventCreate(&e3);
  hipEventCreate(&e4);

  for(i=0; i<5; i++)
  {
   // MemCpy H2D of Kernel 1
   hipMemcpyAsync(d1_a, h1_a, bytes, hipMemcpyHostToDevice, stream1);
   hipMemcpyAsync(d1_b, h1_b, bytes, hipMemcpyHostToDevice, stream1);
   hipEventRecord(e1, stream1);

   // MemCpy H2D of Kernel 2
   hipMemcpyAsync(d2_a, h2_a, bytes, hipMemcpyHostToDevice, stream4);
   hipMemcpyAsync(d2_b, h2_b, bytes, hipMemcpyHostToDevice, stream4);
   hipEventRecord(e3, stream4);

   // Computation Kernel 1
   hipStreamWaitEvent(stream2, e1, 0);
   operation1<<<gridSize, blockSize, 0, stream2>>>(d1_a, d1_b, d1_c, n);
   hipEventRecord(e2, stream2);

   // Computation Kernel 2
   hipStreamWaitEvent(stream5, e3, 0);
   operation2<<<gridSize, blockSize, 0, stream5>>>(d2_a, d2_b, d2_c, n);
   hipEventRecord(e4, stream5);

   // MemCpy D2H of Kernel 1
   hipStreamWaitEvent(stream3, e2, 0);
   hipMemcpyAsync(h1_c, d1_c, bytes, hipMemcpyDeviceToHost, stream3);

   // MemCpy D2H of Kernel 2
   hipStreamWaitEvent(stream6, e4, 0);
   hipMemcpyAsync(h2_c, d2_c, bytes, hipMemcpyDeviceToHost, stream6);
  }

  hipDeviceSynchronize();

  // Destroy events
  hipEventDestroy(e1);
  hipEventDestroy(e2);
  hipEventDestroy(e3);
  hipEventDestroy(e4);

  // Destroy streams
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);
  hipStreamDestroy(stream4);
  hipStreamDestroy(stream5);
  hipStreamDestroy(stream6);

  // Release memory
  hipFree(d1_a);
  hipFree(d1_b);
  hipFree(d1_c);
  hipFree(d2_a);
  hipFree(d2_b);
  hipFree(d2_c);

  hipFree(h1_a);
  hipFree(h1_b);
  hipFree(h1_c);
  hipFree(h2_a);
  hipFree(h2_b);
  hipFree(h2_c);

  return 0;
}
